#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"
#include ""
#include <vector>
#include <string>

#include <stdio.h>
#include "obj_loader.h"
#include "hip/hip_vector_types.h"
#include <iostream>

#include <GL/freeglut.h>

#define DIVID_COUNT 50;
#define TEST_CODE 0;


// 相机参数
float cameraRadius = 0.5f; // 相机到原点的距离
float cameraAngle = 0.0f;  // 相机绕原点旋转的角度

//std::vector<Vertex> vertices; // Your parsed vertices
std::vector<float3> vertices; // Your parsed vertices
std::vector<int3> faces;     // Your parsed faces
std::vector<float3> particles;

float3 hostminVal;
float3 hostmaxVal;

#pragma region morton
#define __all__ __host__ __device__
__all__ unsigned int expandBits(unsigned int v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__all__ unsigned int morton3D(float x, float y, float z)
{
    x = min(max(x * 1024.0f, 0.0f), 1023.0f);
    y = min(max(y * 1024.0f, 0.0f), 1023.0f);
    z = min(max(z * 1024.0f, 0.0f), 1023.0f);
    unsigned int xx = expandBits((unsigned int)x);
    unsigned int yy = expandBits((unsigned int)y);
    unsigned int zz = expandBits((unsigned int)z);
    return xx * 4 + yy * 2 + zz;
}

__all__ unsigned int invertBits(unsigned int v) {
    v = v & 0x49249249u;
    v = (v | (v >> 2)) & 0xC30C30C3u;
    v = (v | (v >> 4)) & 0x0F00F00Fu;
    v = (v | (v >> 8)) & 0xFF0000FFu;
    v = (v | (v >> 16)) & 0x0000FFFFu;
    return v;
}

// Converts a 30-bit Morton code back to 3D float coordinates
__all__ void mortonToFloat(unsigned int morton, float& x, float& y, float& z) {
    unsigned int xx = invertBits(morton >> 0);
    unsigned int yy = invertBits(morton >> 1);
    unsigned int zz = invertBits(morton >> 2);

    // Normalize the values back to the [0,1] range
    x = static_cast<float>(xx) / 1024.0f;
    y = static_cast<float>(yy) / 1024.0f;
    z = static_cast<float>(zz) / 1024.0f;
}
#if TEST_CODE
void test_morton_convert() {
    float x = 0.1;
    float y = 0.9;
    float z = 0.34;
    std::cout << "test morton" << std::endl;
    std::cout << x << "," << y << "," << z << std::endl;
    uint code = morton3D(x, y, z);
    std::cout << "morton:" << code << std::endl;
    float rx = 0;
    float ry = 0;
    float rz = 0;
    mortonToFloat(code, rx, ry, rz);
    std::cout << "re morton" << rx << "," << ry << "," << rz << std::endl;
}
#endif // TEST_CODE


#pragma endregion

#pragma region boudingbox cuda
__device__ float atomicMinFloat(float* addr, float value) {
    int* address_as_i = (int*)addr;
    int old = *address_as_i;
    while (value < __int_as_float(old)) {
        old = atomicCAS(address_as_i, old, __float_as_int(value));
    }
    return __int_as_float(old);
}

__device__ float atomicMaxFloat(float* addr, float value) {
    int* address_as_i = (int*)addr;
    int old = *address_as_i;
    while (value > __int_as_float(old)) {
        old = atomicCAS(address_as_i, old, __float_as_int(value));
    }
    return __int_as_float(old);
}

__global__ void boundingboxKernel(const float3* data, int size, float3 * minVal, float3* maxVal)
{
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    if (i >= size) {
        return;
    }
    
    float3 point = data[i];
    atomicMinFloat(&minVal->x, point.x);
    atomicMinFloat(&minVal->y, point.y);
    atomicMinFloat(&minVal->z, point.z);

    atomicMaxFloat(&maxVal->x, point.x);
    atomicMaxFloat(&maxVal->y, point.y);
    atomicMaxFloat(&maxVal->z, point.z);
}

__global__ void boundingboxKernelV2(const float3* data, int size, float3* minVal, float3* maxVal)
{
    extern __shared__ float3 sharedData[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    if (i >= size) {
        return;
    }

    sharedData[tid] = data[i];

    __syncthreads();

    float3 localMin = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);
    float3 localMax = make_float3(FLT_MIN, FLT_MIN, FLT_MIN);

    if (tid == 0)
    {    
        for (int i = 0; i < blockDim.x && blockIdx.x * blockDim.x + i < size; i++) {
            localMin = fminf(localMin, sharedData[i]);

            localMax = fmaxf(localMax, sharedData[i]);
        }
    }

    if (tid == 0) {
        atomicMinFloat(&minVal->x, localMin.x);
        atomicMinFloat(&minVal->y, localMin.y);
        atomicMinFloat(&minVal->z, localMin.z);

        atomicMaxFloat(&maxVal->x, localMax.x);
        atomicMaxFloat(&maxVal->y, localMax.y);
        atomicMaxFloat(&maxVal->z, localMax.z);
    }

}


void cudaBoudingBox(std::vector<float3>& input)
{

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    float3* data;
    hostminVal = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);
    hostmaxVal = make_float3(FLT_MIN, FLT_MIN, FLT_MIN);
    int size = input.size();
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    float3* deviceMinVal;
    float3* deviceMaxVal;

    hipMalloc(&data, size * sizeof(float3));
    hipMalloc(&deviceMinVal, sizeof(float3));
    hipMalloc(&deviceMaxVal, sizeof(float3));

    hipMemcpy(deviceMinVal, &hostminVal, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(deviceMaxVal, &hostmaxVal, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(data, input.data(), size * sizeof(float3), hipMemcpyHostToDevice);

    hipEventRecord(start);
#if 1
    boundingboxKernel << < gridSize, blockSize >> > (data, size, deviceMinVal, deviceMaxVal);
#else
    const size_t smSz = blockSize * sizeof(float3);
    boundingboxKernelV2 << < gridSize, blockSize, smSz >> > (data, size, deviceMinVal, deviceMaxVal);
#endif
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(&hostminVal, deviceMinVal, sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(&hostmaxVal, deviceMaxVal, sizeof(float3), hipMemcpyDeviceToHost);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Kernel execution time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Bounding Box Min: (" << hostminVal.x << ", " << hostminVal.y << ", " << hostminVal.z << ")\n";
    std::cout << "Bounding Box Max: (" << hostmaxVal.x << ", " << hostmaxVal.y << ", " << hostmaxVal.z << ")\n";

    hipFree(data);
    hipFree(deviceMinVal);
    hipFree(deviceMaxVal);
}

#pragma endregion

#pragma region boudingbox cpu
void cpuBoundingBox() {
    // 初始化边界框的最小和最大值
    float min_x = vertices[0].x;
    float min_y = vertices[0].y;
    float min_z = vertices[0].z;
    float max_x = vertices[0].x;
    float max_y = vertices[0].y;
    float max_z = vertices[0].z;

    // 遍历所有顶点，更新最小和最大值
    for (const auto& vertex : vertices) {
        min_x = std::fmin(min_x, vertex.x);
        min_y = std::fmin(min_y, vertex.y);
        min_z = std::fmin(min_z, vertex.z);
        max_x = std::fmax(max_x, vertex.x);
        max_y = std::fmax(max_y, vertex.y);
        max_z = std::fmax(max_z, vertex.z);
    }

    // 打印边界框信息
    std::cout << "Bounding Box:" << std::endl;
    std::cout << "Min X: " << min_x << std::endl;
    std::cout << "Max X: " << max_x << std::endl;
    std::cout << "Min Y: " << min_y << std::endl;
    std::cout << "Max Y: " << max_y << std::endl;
    std::cout << "Min Z: " << min_z << std::endl;
    std::cout << "Max Z: " << max_z << std::endl;
}
#pragma endregion

#pragma region all particles within bouding box
#define GENRATE_BY_CUDA 1
__device__ __host__ float divid_length(float3 min, float3 maxIn) {
    float3 range = maxIn - min;
    float maxv = max(range.x, range.y);
    maxv = max(maxv, range.z);
    return maxv / DIVID_COUNT;
}

__global__ void cuda_generate_particle_within_bouding_box(float3* particlebuffer, float3 min_p, float divid_len, int x_len, int y_len, int z_len) {
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    int index = i;
    int count = x_len * y_len * z_len;
    if (i >= count) {
        return;
    }
    int zid = i / (x_len * y_len);
    i = i - zid * (x_len * y_len);
    int yid = i / (x_len);
    i = i - yid * (x_len);
    int xid = i;

    particlebuffer[index] = min_p + make_float3(divid_len * xid, divid_len * yid, divid_len * zid);
}

void generate_particle_within_bouding_box(float3 min_p, float3 max_p) {
    float divid_len = divid_length(min_p, max_p);

    std::cout << "divid len:" << divid_len << std::endl;
    if (divid_len == 0) {
        return;
    }
#if GENRATE_BY_CUDA
    float3* particlebuffer;
    float3 dif_p = max_p - min_p;
    int x_len = dif_p.x / divid_len+1;
    int y_len = dif_p.y / divid_len+1;
    int z_len = dif_p.z / divid_len+1;
    int count = x_len * y_len * z_len;
    std::cout << "len:" << x_len << ", " << y_len << ", " << z_len << std::endl;
    hipMalloc(&particlebuffer, count * sizeof(float3));
    int blocksize = 256;
    int gridcount = (count + blocksize - 1) / blocksize;
    cuda_generate_particle_within_bouding_box << <gridcount, blocksize  >> > (particlebuffer, min_p, divid_len, x_len, y_len, z_len);

    particles.resize(count);
    hipMemcpy(particles.data(), particlebuffer, count * sizeof(float3), hipMemcpyDeviceToHost);
    hipFree(particlebuffer);

#else
    for (float x = min_p.x; x <= max_p.x; x += divid_len) {
        for (float y = min_p.y; y <= max_p.y; y += divid_len) {
            for (float z = min_p.z; z <= max_p.z; z += divid_len) {
                particles.push_back(make_float3(x, y, z));
            }
        }
    }
#endif
}
#pragma endregion

#pragma region paricle from triangles

#define MAX_PARTICLES_COUNT 70000
__device__ __inline__ void put_data(float3* data, int* current_index, float3 value) {
    int returnIndex = atomicAdd(current_index, 1);
    if (returnIndex >= MAX_PARTICLES_COUNT) {
        return;
    }
    data[returnIndex] = value;

}

__device__ __inline__ int3 toIndex(float3 min, float3 val, float divid_len) {
    float3 dif = val - min;
    return make_int3(
        (int)(dif.x / divid_len),
        (int)(dif.y / divid_len),
        (int)(dif.z / divid_len)
    );
}

__device__ __inline__ float3 toPosition(float3 min, int x, int y, int z, float divid_len) {
    return make_float3(
        min.x + x * divid_len,
        min.y + y * divid_len,
        min.z + z * divid_len
    );
}

__device__ __inline__ float3 toPosition(float3 min, int3 val, float divid_len) {
    return make_float3(
        min.x + val.x * divid_len,
        min.y + val.y * divid_len,
        min.z + val.z * divid_len
    );
}

__device__ __inline__ float3 toGrid(float3 min, float3 val, float divid_len) {
    float3 dif = val - min;
    return make_float3(
        min.x + (uint)(dif.x / divid_len) * divid_len,
        min.y + (uint)(dif.y / divid_len) * divid_len,
        min.z + (uint)(dif.z / divid_len) * divid_len
    );
}

__global__ void cuda_generate_particles_from_triangles(float3* vertices, int3* faces, float3* particles, size_t face_count, float3 min, float divid_len, int* particle_count) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    if (index >= face_count) {
        return;
    }
    int3 face = faces[index];
    float3 v1 = vertices[face.x];
    float3 v2 = vertices[face.y];
    float3 v3 = vertices[face.z];
#if 0 // vertice grid position
    put_data(particles, particle_count, toGrid(min, v1, divid_len));
    put_data(particles, particle_count, toGrid(min, v2, divid_len));
    put_data(particles, particle_count, toGrid(min, v3, divid_len));
#endif

    float3 minv = v1;
    minv = fminf(minv, v2);
    minv = fminf(minv, v3);
    float3 maxv = v1;
    maxv = fmaxf(maxv, v2);
    maxv = fmaxf(maxv, v3);
    int3 lowIndex = toIndex(min, minv, divid_len);
    int3 highIndex = toIndex(min, maxv, divid_len);

    for (int x = lowIndex.x; x <= highIndex.x; x++) {
        for (int y = lowIndex.y; y <= highIndex.y; y++) {
            for (int z = lowIndex.z; z <= highIndex.z; z++) {
                put_data(particles, particle_count, toPosition(min, x, y, z, divid_len));
            }
        }
    }

}

void generate_particles_from_triangles() {
    float divid_len = divid_length(hostminVal, hostmaxVal);
    
    float3* deviceVertices;
    int3* deviceFaces;
    float3* deviceParticles;
    int* particle_count;

    hipMalloc(&deviceVertices, vertices.size() * sizeof(float3));
    hipMalloc(&deviceFaces, faces.size() * sizeof(int3));
    hipMalloc(&deviceParticles, MAX_PARTICLES_COUNT * sizeof(float3));
    hipMalloc(&particle_count, sizeof(int));

    hipMemcpy(deviceVertices, vertices.data(), vertices.size() * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(deviceFaces, faces.data(), faces.size() * sizeof(int3), hipMemcpyHostToDevice);
    int host_particle_count = 0;
    hipMemcpy(&particle_count, &host_particle_count, sizeof(int), hipMemcpyHostToDevice);

    size_t face_count = faces.size();
    size_t blocksize = 256;
    size_t gridcount = (face_count + blocksize - 1) / blocksize;

    cuda_generate_particles_from_triangles << <gridcount, blocksize >> > (deviceVertices, deviceFaces, deviceParticles, face_count, hostminVal, divid_len, particle_count);
    hipMemcpy(&host_particle_count, particle_count, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "getting particle count:" << host_particle_count << std::endl;
    host_particle_count = min(host_particle_count, MAX_PARTICLES_COUNT);

    particles.resize(host_particle_count);
    hipMemcpy(particles.data(), deviceParticles, host_particle_count * sizeof(float3), hipMemcpyDeviceToHost);
    
    hipFree(deviceVertices);
    hipFree(deviceFaces);
    hipFree(deviceParticles);
    //hipFree(deviceCount);
}

#pragma endregion

#pragma region freeglut render and interact
void reshape(int width, int height) {
    // 防止除以零
    if (height == 0) {
        height = 1;
    }

    // 设置视口
    glViewport(0, 0, width, height);

    // 设置投影矩阵
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // 设置视角
    gluPerspective(45.0, (float)width / (float)height, 0.1, 100.0);
}

void display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    // 设置相机位置
    float cameraX = cameraRadius * std::cos(cameraAngle);
    float cameraY = cameraRadius * std::sin(cameraAngle);
    gluLookAt(cameraX, 0.5, cameraY,  // 相机位置
        0.0, 0.0, 0.0,          // 观察点
        0.0, 1.0, 0.0);         // 上方向
#if 0
    // triangle
    glBegin(GL_TRIANGLES);
    for (const auto& face : faces) {
        glColor3f(1.0, 0.0, 0.0);
        glVertex3f(vertices[face.x].x, vertices[face.x].y, vertices[face.x].z);
        glColor3f(0.0, 1.0, 0.0);
        glVertex3f(vertices[face.y].x, vertices[face.y].y, vertices[face.y].z);
        glColor3f(0.0, 0.0, 1.0);
        glVertex3f(vertices[face.z].x, vertices[face.z].y, vertices[face.z].z);
    }
    glEnd();
#endif

    // point
    //glPointSize(3.0f);
    glBegin(GL_POINTS);
    for (const auto& vertice : particles) {
        glColor3f(1.0f, 1.0f, 1.0f); // White color for particles
        glVertex3f(vertice.x, vertice.y, vertice.z);
    }
    glEnd();

#if TEST_CODE
    // 检查 OpenGL 错误状态
    GLenum error = glGetError();
    if (error != GL_NO_ERROR) {
        std::cerr << "OpenGL error: " << gluErrorString(error) << std::endl;
    }
    else {
        std::cout << "sucess" << std::endl;
    }
#endif

    glutSwapBuffers();
}
void specialKeys(int key, int x, int y) {
    const float rotationSpeed = 0.01f;
    if (key == GLUT_KEY_LEFT) {
        cameraAngle -= rotationSpeed;
    }
    else if (key == GLUT_KEY_RIGHT) {
        cameraAngle += rotationSpeed;
    }
    glutPostRedisplay();
}

void run(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(640, 480);
    glutCreateWindow("Polygon Viewer");
    glEnable(GL_DEPTH_TEST);
    // 设置重塑回调
    glutReshapeFunc(reshape);
    // Set display callback
    glutDisplayFunc(display);
    glutSpecialFunc(specialKeys);

    // Your other initialization code

    // Enter GLUT main loop
    glutMainLoop();
}
#pragma endregion

int main(int argc, char** argv)
{
#if TEST_CODE
    test_morton_convert();
#endif
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    const char* objFilename = "D:\\Download\\bunny.obj";

    readObjFile(objFilename, vertices, faces);
    std::cout << "vertices:" << vertices.size() << ", faces:" << faces.size() << std::endl;
    cudaBoudingBox(vertices);
#if 0
    generate_particle_within_bouding_box(hostminVal, hostmaxVal);
#else
    generate_particles_from_triangles();
#endif

    cpuBoundingBox();
    run(argc, argv);
    return 0;
}